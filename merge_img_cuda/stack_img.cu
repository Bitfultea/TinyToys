#include "hip/hip_runtime.h"
#include <hip/hip_runtime.h>
#include <stdio.h>

#include <iostream>
#include <opencv2/highgui.hpp>
#include <string>

#include "hip/hip_runtime.h"
#include "opencv2/imgproc/imgproc.hpp"

#define BLOCK_SIZE 16

__global__ void stack_maxid(unsigned char *srcImage_list,
                            unsigned char *maxImage,
                            unsigned char *max_id_Image,
                            unsigned int width,
                            unsigned int height,
                            unsigned int img_num) {
    int x = blockIdx.x * blockDim.x + threadIdx.x;
    int y = blockIdx.y * blockDim.y + threadIdx.y;

    int img_size = width * height;
    int max = 0;
    int idx = 0;
    for (int i = 0; i < img_num; i++) {
        if (srcImage_list[(i * img_size + y * width + x)] > max) {
            max = srcImage_list[(i * img_size + y * width + x)];
            idx = i;
        }
        // max = i * 200 + 10;
    }

    maxImage[(y * width + x)] = max;
    max_id_Image[(y * width + x)] = idx;
}

__global__ void stack_max(unsigned char *srcImage_list,
                          unsigned char *maxImage,
                          unsigned int width,
                          unsigned int height,
                          unsigned int img_num) {
    int x = blockIdx.x * blockDim.x + threadIdx.x;
    int y = blockIdx.y * blockDim.y + threadIdx.y;

    int img_size = width * height;
    int max = 0;
    for (int i = 0; i < img_num; i++) {
        if (srcImage_list[(i * img_size + y * width + x)] > max) {
            max = srcImage_list[(i * img_size + y * width + x)];
        }
        // max = i * 200 + 10;
    }

    maxImage[(y * width + x)] = max;
}

__global__ void stack_average(unsigned char *srcImage_list,
                              unsigned char *aveImage,
                              unsigned int width,
                              unsigned int height,
                              unsigned int img_num) {
    int x = blockIdx.x * blockDim.x + threadIdx.x;
    int y = blockIdx.y * blockDim.y + threadIdx.y;

    int img_size = width * height;
    float average = 0;
    for (int i = 0; i < img_num; i++) {
        average += srcImage_list[(i * img_size + y * width + x)];
    }
    average = average / img_num;
    aveImage[(y * width + x)] = average;
}

extern "C" void stackimg_GPU_wrapper(int stack_mode,
                                     const std::vector<cv::Mat> &input,
                                     cv::Mat &output) {
    if (stack_mode == 0 || stack_mode == 1 || stack_mode == 2) {
        // Start time
        hipEvent_t start, stop;
        cv::Mat max_debug(output.size(), output.type());

        hipEventCreate(&start);
        hipEventCreate(&stop);

        unsigned int img_num = input.size();
        const int inputSize = input[0].cols * input[0].rows * img_num;
        const int outputSize = output.cols * output.rows;
        unsigned char *d_input, *d_output, *d_max;

        // Allocate device memory
        hipMalloc<unsigned char>(&d_input, inputSize);
        hipMalloc<unsigned char>(&d_output, outputSize);
        if (stack_mode == 0) hipMalloc<unsigned char>(&d_max, outputSize);

        // Copy data from OpenCV input image to device memory
        // hipMemcpy(d_input, &input, inputSize, hipMemcpyHostToDevice);
        for (int i = 0; i < input.size(); i++) {
            hipMemcpy(d_input + i * input[0].total(), input[i].ptr(),
                       input[i].total(), hipMemcpyHostToDevice);
        }

        // Specify block size
        const dim3 block(BLOCK_SIZE, BLOCK_SIZE);

        // Calculate grid size to cover the whole image
        const dim3 grid((output.cols + block.x - 1) / block.x,
                        (output.rows + block.y - 1) / block.y);

        // Start time
        hipEventRecord(start);

        // Run Sobel Edge Detection Filter kernel on CUDA
        if (stack_mode == 0) {
            stack_maxid<<<grid, block>>>(d_input, d_max, d_output, output.cols,
                                         output.rows, img_num);
        } else if (stack_mode == 1) {
            stack_max<<<grid, block>>>(d_input, d_output, output.cols,
                                       output.rows, img_num);
        } else if (stack_mode == 2) {
            stack_average<<<grid, block>>>(d_input, d_output, output.cols,
                                           output.rows, img_num);
        }

        // Stop time
        hipEventRecord(stop);

        // Copy data from device memory to output image
        hipMemcpy(output.ptr(), d_output, outputSize, hipMemcpyDeviceToHost);
        if (stack_mode == 0) {
            hipMemcpy(max_debug.ptr(), d_max, outputSize,
                       hipMemcpyDeviceToHost);
        }

        // Free the device memory
        hipFree(d_input);
        hipFree(d_output);
        hipFree(d_max);

        hipEventSynchronize(stop);
        float milliseconds = 0;

        if (stack_mode == 0) {
            max_debug.convertTo(max_debug, CV_32F, 1.0 / 255, 0);
            max_debug *= 255;
            cv::imwrite("max_debug.jpg", max_debug);
        }

        // Calculate elapsed time in milisecond
        hipEventElapsedTime(&milliseconds, start, stop);
        std::cout << "\nMerge Processing time on GPU (ms): " << milliseconds
                  << "\n";
    } else {
        return;  // error
    }
}