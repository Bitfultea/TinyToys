#include "hip/hip_runtime.h"
#include <hip/hip_runtime.h>
#include <stdio.h>

#include <iostream>
#include <opencv2/highgui.hpp>
#include <string>

#include "hip/hip_runtime.h"
#include "opencv2/imgproc/imgproc.hpp"

#define BLOCK_SIZE 16

__global__ void stack_maxid(unsigned char *srcImage_list,
                            unsigned char *maxImage,
                            unsigned char *max_id_Image,
                            unsigned int width,
                            unsigned int height) {
    int x = blockIdx.x * blockDim.x + threadIdx.x;
    int y = blockIdx.y * blockDim.y + threadIdx.y;

    int img_size = width * height;
    // int img_num = sizeof(*srcImage_list) / img_size;
    int img_num = 3;
    int max = 0;
    int idx = 0;
    for (int i = 0; i < img_num; i++) {
        if (srcImage_list[(i * img_size + y * width + x)] > max) {
            max = srcImage_list[(i * img_size + y * width + x)];
            idx = i;
        }
        // max = i * 200 + 10;
    }

    maxImage[(y * width + x)] = max;
    max_id_Image[(y * width + x)] = idx;
}

__global__ stack_max(unsigned char *srcImage_list,
                     unsigned char *maxImage,
                     unsigned int width,
                     unsigned int height) {
    int x = blockIdx.x * blockDim.x + threadIdx.x;
    int y = blockIdx.y * blockDim.y + threadIdx.y;

    int img_size = width * height;
    // int img_num = sizeof(*srcImage_list) / img_size;
    int img_num = 3;
    int max = 0;
    for (int i = 0; i < img_num; i++) {
        if (srcImage_list[(i * img_size + y * width + x)] > max) {
            max = srcImage_list[(i * img_size + y * width + x)];
        }
        // max = i * 200 + 10;
    }

    maxImage[(y * width + x)] = max;
}

__global__ stack_average(unsigned char *srcImage_list,
                         unsigned char *aveImage,
                         unsigned int width,
                         unsigned int height) {
    int x = blockIdx.x * blockDim.x + threadIdx.x;
    int y = blockIdx.y * blockDim.y + threadIdx.y;

    int img_size = width * height;
    int img_num = 3;
    float average = 0;
    for (int i = 0; i < img_num; i++) {
        if (srcImage_list[(i * img_size + y * width + x)] > max) {
            average += srcImage_list[(i * img_size + y * width + x)];
        }
        // max = i * 200 + 10;
    }
    average = average / 3.0;
    aveImage[(y * width + x)] = max;
}

extern "C" void get_max_id_GPU_wrapper(const std::vector<cv::Mat> &input,
                                       cv::Mat &output) {
    hipEvent_t start, stop;
    cv::Mat max_debug(output.size(), output.type());
    hipEventCreate(&start);
    hipEventCreate(&stop);

    const int inputSize = input[0].cols * input[0].rows * input.size();
    const int outputSize = output.cols * output.rows;
    unsigned char *d_input, *d_output, *d_max;
    // unsigned char *d_output, *d_max;

    // Allocate device memory
    // cv::Mat *d_mats;
    // hipMalloc((void **)&d_mats, mats.size() * sizeof(cv::Mat));
    hipMalloc<unsigned char>(&d_input, inputSize);
    hipMalloc<unsigned char>(&d_output, outputSize);
    hipMalloc<unsigned char>(&d_max, outputSize);

    // Copy data from OpenCV input image to device memory
    // hipMemcpy(d_input, &input, inputSize, hipMemcpyHostToDevice);
    for (int i = 0; i < input.size(); i++) {
        hipMemcpy(d_input + i * input[0].total(), input[i].ptr(),
                   input[i].total(), hipMemcpyHostToDevice);
    }

    // Specify block size
    const dim3 block(BLOCK_SIZE, BLOCK_SIZE);

    // Calculate grid size to cover the whole image
    const dim3 grid((output.cols + block.x - 1) / block.x,
                    (output.rows + block.y - 1) / block.y);

    // Start time
    hipEventRecord(start);

    // Run Sobel Edge Detection Filter kernel on CUDA
    stack_maxid<<<grid, block>>>(d_input, d_max, d_output, output.cols,
                                 output.rows);

    // Stop time
    hipEventRecord(stop);

    // Copy data from device memory to output image
    hipMemcpy(output.ptr(), d_output, outputSize, hipMemcpyDeviceToHost);
    hipMemcpy(max_debug.ptr(), d_max, outputSize, hipMemcpyDeviceToHost);

    // Free the device memory
    hipFree(d_input);
    hipFree(d_output);
    hipFree(d_max);

    hipEventSynchronize(stop);
    float milliseconds = 0;

    max_debug.convertTo(max_debug, CV_32F, 1.0 / 255, 0);
    max_debug *= 255;
    cv::imwrite("max_debug.jpg", max_debug);

    // Calculate elapsed time in milisecond
    hipEventElapsedTime(&milliseconds, start, stop);
    std::cout << "\nMerge Processing time on GPU (ms): " << milliseconds
              << "\n";
}